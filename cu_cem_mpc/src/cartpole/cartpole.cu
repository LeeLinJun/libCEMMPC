#include "hip/hip_runtime.h"
#include "cartpole/cartpole.h"
#include "cuda_dep.h"

#define I 10
#define L 2.5
#define M 10
#define m 5
#define g 9.8

#define DT  2e-3
#define PI  3.141592654f

#define MIN_X -30
#define MAX_X 30
#define MIN_V -40
#define MAX_V 40
#define MIN_W -2
#define MAX_W 2

#define DIM_STATE 4
#define DIM_CONTROL 1
#define STATE_X 0
#define STATE_V 1
#define STATE_THETA 2
#define STATE_W 3

namespace cartpole{

    __global__ 
    void set_statistics(double* d_mean_time, const double mean_time, double* d_mean_control, const double mean_control, 
        double* d_std_control, const double std_control, double* d_std_time, const double std_time, int N){
        unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;// 0~NT
        if (id < N){
            d_mean_time[id] = mean_time;
            d_mean_control[id] = mean_control;
            d_std_control[id] = std_control;
            d_std_time[id] = std_time;
        }
    }

    __global__
    void set_start_state(double* temp_state, const double x, const double v, const double theta, const double w, const int N){
        unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
        if (id < N){
            temp_state[STATE_X + id*DIM_STATE] = x;
            temp_state[STATE_V + id*DIM_STATE] = v;
            temp_state[STATE_THETA + id*DIM_STATE] = theta;
            temp_state[STATE_W + id*DIM_STATE] = w;
        }
    }

    __global__ 
    void sampling(double* control, double* time, double* mean_control, double* mean_time, double* std_control, double* std_time, const int NS, const int NT){
        unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;// each sample
        hiprandState state;
        hiprand_init(clock(), id, 0, &state);
        if(id < NS){
            for(unsigned int t = 0; t < NT; t++){
                control[t * NS + id] = std_control[t] * hiprand_normal(&state) + mean_control[t];
                // hiprand_init(clock(), id + t, 0, &state);
                time[t * NS + id] = std_time[t] * hiprand_normal(&state) + mean_time[t];
                // printf("%f, %f\n",control[t * NS + id],time[t * NS + id]);
                if(time[t * NS + id] < DT){
                    time[t * NS + id] = 0;
                }
            }
            
        }
    }

    __global__
    void propagate(double* temp_state, double* control, double* time, double* deriv, const int t_step, const int N){
        unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
        if (id < N){
            double t = time[t_step*N + id];
            if (t < 0){
                t = 0;
            }
            int num_step = (t + 0.1*DT) / DT;
            double _a = control[id + t_step * N];

            for(unsigned int i = 0; i < num_step; i++){
                // update derivs
                double _v = temp_state[STATE_V + id*DIM_STATE];
                double _w = temp_state[STATE_W + id*DIM_STATE];
                double _theta = temp_state[STATE_THETA + id*DIM_STATE];
                double mass_term = 1.0 / ((M + m)*(I + m * L * L) - m * m * L * L * cos(_theta) * cos(_theta));
                deriv[STATE_X + id*DIM_STATE] = _v;
                deriv[STATE_THETA + id*DIM_STATE] = _w;
                deriv[STATE_V + id*DIM_STATE] = ((I + m * L * L)*(_a + m * L * _w * _w * sin(_theta)) + m * m * L * L * cos(_theta) * sin(_theta) * g) * mass_term;
                deriv[STATE_W + id*DIM_STATE] = ((-m * L * cos(_theta))*(_a + m * L * _w * _w * sin(_theta))+(M + m)*(-m * g * L * sin(_theta))) * mass_term;
                // update states
                temp_state[STATE_X + id*DIM_STATE] += DT * deriv[STATE_X + id*DIM_STATE];
                temp_state[STATE_THETA + id*DIM_STATE] += DT * deriv[STATE_THETA + id*DIM_STATE];
                temp_state[STATE_V + id*DIM_STATE] += DT * deriv[STATE_V + id*DIM_STATE];
                temp_state[STATE_W + id*DIM_STATE] += DT * deriv[STATE_W + id*DIM_STATE];
                // enforce bounds
                if (temp_state[STATE_THETA + id*DIM_STATE] > PI){
                    temp_state[STATE_THETA + id*DIM_STATE] -= 2 * PI;
                }else if(temp_state[STATE_THETA + id*DIM_STATE] < -PI){
                    temp_state[STATE_THETA + id*DIM_STATE] += 2 * PI;
                }
                if (temp_state[STATE_V + id*DIM_STATE] > MAX_V){
                    temp_state[STATE_V + id*DIM_STATE] = MAX_V;
                }else if(temp_state[STATE_V + id*DIM_STATE] < MIN_V){
                    temp_state[STATE_V + id*DIM_STATE] = MIN_V;
                }
                if (temp_state[STATE_W + id*DIM_STATE] > MAX_W){
                    temp_state[STATE_W + id*DIM_STATE] = MAX_W;
                }else if(temp_state[STATE_W + id*DIM_STATE] < MIN_W){
                    temp_state[STATE_W + id*DIM_STATE] = MIN_W;
                }
            }        
        }
    }

    __global__
    void get_loss(double* temp_state, double* loss, const int N, double goal0, double goal1, double goal2, double goal3){
        unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;// each id is a sample
        if (id < N){
            loss[id] = sqrt((temp_state[id*DIM_STATE + STATE_X]-goal0) * (temp_state[id*DIM_STATE + STATE_X]-goal0)\
                + 0.5*(temp_state[id*DIM_STATE + STATE_V]-goal1) * (temp_state[id*DIM_STATE + STATE_V]-goal1)\
                + (temp_state[id*DIM_STATE + STATE_THETA]-goal2) * (temp_state[id*DIM_STATE + STATE_THETA]-goal2)\
                + 0.5 * (temp_state[id*DIM_STATE + STATE_W]-goal3) * (temp_state[id*DIM_STATE + STATE_W]-goal3));
        }
    }

    __global__
    void update_statistics(double* control, double* time, double* mean_control, double* mean_time, double* std_control, double* std_time,
        int* loss_ind, double* loss, int NS, int NT, int N_ELITE, double* best_ut){
        unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
        if(id < NT){
            double sum_control = 0., sum_time = 0., ss_control = 0., ss_time = 0.;
            for(int i = 0; i < N_ELITE; i++){
                sum_control += control[loss_ind[i] + id*NS];
                ss_control += control[loss_ind[i] + id*NS] * control[loss_ind[i] + id*NS];
                sum_time += time[loss_ind[i] + id*NS];
                ss_time += time[loss_ind[i] + id*NS] * time[loss_ind[i] + id*NS];
            }
            // printf("%f,%f\n",ss_control, ss_time);
            mean_control[id] = sum_control / N_ELITE;
            mean_time[id] = sum_time / N_ELITE;
            std_control[id] = sqrt(ss_control / N_ELITE - mean_control[id] * mean_control[id]);
            std_time[id] = sqrt(ss_time / N_ELITE - mean_time[id] * mean_time[id]);
            best_ut[id] = control[loss_ind[0] + id*NS];
            best_ut[id + NT] = time[loss_ind[0] + id*NS];
            
        }
    }

    Cartpole::Cartpole(int ns, int n_elete, int nt, int block_size):NS(ns),
    N_ELITE(n_elete), NT(nt), BLOCK_SIZE(block_size){
        printf("setup...\n");
        best_ut = (double*) malloc(NT * 2 * sizeof(double));
        hipMalloc(&d_best_ut, NT * 2 * sizeof(double)); 
        // temp_state = (double*) malloc(NS * DIM_STATE * sizeof(double));
        hipMalloc(&d_temp_state, NS * DIM_STATE * sizeof(double)); 
        hipMalloc(&d_deriv, NS * DIM_STATE * sizeof(double));
        hipMalloc(&d_control, NS * NT * DIM_CONTROL * sizeof(double));
        hipMalloc(&d_time, NS * NT * sizeof(double));
        // for sampling
        hipMalloc(&d_mean_time, NT * sizeof(double)); 
        hipMalloc(&d_mean_control, NT* sizeof(double));
        hipMalloc(&d_std_control, NT * sizeof(double));
        hipMalloc(&d_std_time, NT * sizeof(double));
        // for cem
        hipMalloc(&d_loss, NS * sizeof(double)); 
        hipMalloc(&d_loss_ind, NS * sizeof(int)); 
        loss_ind = (int*) malloc(NS * sizeof(int));
        memset(loss_ind, 0, NS  * sizeof(int));
        printf("done, execution:\n");

    }

    void Cartpole::cem(double* start, double* goal){
        auto begin = std::chrono::system_clock::now();
        thrust::device_ptr<double> time_ptr(d_time);
        thrust::device_ptr<double> control_ptr(d_control);
        thrust::device_ptr<double> loss_ptr(d_loss);
        thrust::device_ptr<int> loss_ind_ptr(d_loss_ind);
        //init mean
        set_statistics<<<(NT+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_mean_time, 0, d_mean_control, 0.0, d_std_control, 500, d_std_time, 5e-2, NT);

        double min_loss = 1e5;
        double tmp_min_loss = 2e5;
        auto init_end = std::chrono::system_clock::now();

        for(unsigned int it = 0; it < 100; it ++){
            set_start_state<<<(NS+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_temp_state, start[0], start[1], start[2], start[3], NS);

            sampling<<<(NS+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_control, d_time, d_mean_control, d_mean_time, d_std_control, d_std_time, NS, NT);
            for(unsigned int t_step = 0; t_step < NT; t_step++){
                propagate<<<(NS+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_temp_state, d_control, d_time, d_deriv, t_step, NS);
            }
            get_loss<<<(NS+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_temp_state, d_loss, NS, goal[0], goal[1], goal[2], goal[3]);
            thrust::sequence(loss_ind_ptr, loss_ind_ptr+NS);
            thrust::sort_by_key(loss_ptr, loss_ptr + NS, loss_ind_ptr);

            update_statistics<<<NT, BLOCK_SIZE>>>(d_control, d_time, d_mean_control, d_mean_time, d_std_control, d_std_time,
                thrust::raw_pointer_cast(loss_ind_ptr),  thrust::raw_pointer_cast(loss_ptr), NS, NT, N_ELITE, d_best_ut);

            hipMemcpy(&tmp_min_loss, thrust::raw_pointer_cast(loss_ptr), sizeof(double), hipMemcpyDeviceToHost);

            if(tmp_min_loss < min_loss){
                min_loss = tmp_min_loss;
                hipMemcpy(best_ut, d_best_ut, 2 * NT * sizeof(double), hipMemcpyDeviceToHost);

            }
            printf("%f,\t%f\n", tmp_min_loss, min_loss);

            if(min_loss < 1e-1){
                break;
            }
        }
        auto done = std::chrono::system_clock::now();
        printf("done\n");

        printf("control = [");
        for(unsigned int it = 0; it < NT; it ++){
            printf("%f,", best_ut[it]);
        }
        printf("]\ntime = [");
        for(unsigned int it = 0; it < NT; it ++){
            printf("%f,", best_ut[it+NT]);
        }
        printf("]\n");


        auto duration_init = std::chrono::duration_cast<std::chrono::microseconds>(init_end-begin);
        auto duration_exec = std::chrono::duration_cast<std::chrono::microseconds>(done-init_end);
        printf("init:%f\nexec:%f\n",double(duration_init.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den,
            double(duration_exec.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den);
    
        // return d_control;
    }

    Cartpole::~Cartpole(){
        hipFree(d_temp_state);
        hipFree(d_control);
        hipFree(d_deriv);
        hipFree(d_time);
        hipFree(d_mean_time);
        hipFree(d_mean_control);
        hipFree(d_std_control);
        hipFree(d_std_time);
        // free(temp_state);
    }
}